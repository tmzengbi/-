#ifndef __CUDACC__
#define __CUDACC__
#endif

#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void solve(float *A, float *B, float *C, int N, int M, int K) {
    #define idA (i * M + k) 
    #define idB (k * K + j)
    #define idC (i * K + j)

    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    int i = threadId;
    if(i >= N) return;
    for(int k = 0; k < M; ++ k) {
        float tmp = A[idA];
        for(int j = 0; j < K; ++ j)
            C[idC] += tmp * B[idB];
    }
    #undef idA
    #undef idB
    #undef idC
}

int main(int argc, char *argv[]) {
    if(argc != 2) {
        fprintf(stderr, "Number of block size needed");
        exit(-1);
    }
    const int THREADS_PER_BLOCK = strtol(argv[1], NULL, 10);
    float *A, *B, *C;
    int N, M, K;
    FILE *fp;
    if((fp = fopen("a.in", "r")) == NULL) {
        perror("cannot open file");
        exit(-1);
    }
    fread(&N, 1, sizeof(int), fp);
    fread(&M, 1, sizeof(int), fp);
    fread(&K, 1, sizeof(int), fp);
    A = (float*) malloc (sizeof(float) * N * M);
    B = (float*) malloc (sizeof(float) * M * K);
    C = (float*) malloc (sizeof(float) * N * K);
    fread(A, sizeof(float), N * M, fp);
    fread(B, sizeof(float), M * K, fp);
    #ifdef DEBUG
    printf("Matrix A:\n");
    for(int i = 0; i < N * M; ++ i) {
        printf("%f ",A[i]);
        if(i % M == M - 1)
            printf("\n");
    }
    printf("Matrix B:\n");
    for(int i = 0; i < M * K; ++ i) {
        printf("%f ",B[i]);
        if(i % K == K - 1)
            printf("\n");
    }
    #endif
    float *d_A, *d_B, *d_C;
    hipMalloc((void**) &d_A, sizeof(float) * N * M);
    hipMalloc((void**) &d_B, sizeof(float) * M * K);
    hipMalloc((void**) &d_C, sizeof(float) * N * K);
    printf("start copying data from Host to Device\n");
    hipMemcpy(d_A, A, sizeof(float) * N * M, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeof(float) * M * K, hipMemcpyHostToDevice);
    hipMemset(d_C, 0, sizeof(float) * N * K);
    printf("start calculating\n");
    hipEvent_t e_start, e_stop;
    hipEventCreate(&e_start);
    hipEventCreate(&e_stop);

    hipEventRecord(e_start, 0);
    solve<<<(N + THREADS_PER_BLOCK -1) / THREADS_PER_BLOCK ,THREADS_PER_BLOCK>>>(d_A, d_B, d_C, N, M, K);
    hipEventRecord(e_stop, 0);

    hipEventSynchronize(e_stop);
    float elapsedtime;
    hipEventElapsedTime(&elapsedtime, e_start, e_stop);
    printf("finish calculating, costing %f\n", elapsedtime);

    printf("start copying data from Device to Host\n");
    hipMemcpy(C, d_C, sizeof(float) * N * K,hipMemcpyDeviceToHost);
    printf("Normal execution\n");
    #ifdef DEBUG
    for(int i = 0; i < N * K; ++ i) {
        printf("%f ",C[i]);
        if(i % K == K - 1)
            printf("\n");
    }
    #endif
    hipError_t err = hipGetLastError();
    printf("%s\n",hipGetErrorString(err));
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);
}
