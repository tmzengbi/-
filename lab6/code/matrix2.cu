#ifndef __CUDACC__
#define __CUDACC__
#endif

#include <cstdio>
#include <cstring>
#include <hip/hip_runtime.h>
#include <hipblas.h>
using namespace std;

int main(int argc, char *argv[]) {
    float *A, *B, *C;
    const float alpha = 1;
    const float beta = 0;
    int N, M, K;
    FILE *fp;
    if((fp = fopen("a.in", "r")) == NULL) {
        perror("cannot open file");
        exit(-1);
    }
    fread(&N, 1, sizeof(int), fp);
    fread(&M, 1, sizeof(int), fp);
    fread(&K, 1, sizeof(int), fp);
    A = (float*) malloc (sizeof(float) * N * M);
    B = (float*) malloc (sizeof(float) * M * K);
    C = (float*) malloc (sizeof(float) * N * K);
    fread(A, sizeof(float), N * M, fp);
    fread(B, sizeof(float), M * K, fp);
    #ifdef DEBUG
    printf("Matrix A:\n");
    for(int i = 0; i < N * M; ++ i) {
        printf("%f ",A[i]);
        if(i % M == M - 1)
            printf("\n");
    }
    printf("Matrix B:\n");
    for(int i = 0; i < M * K; ++ i) {
        printf("%f ",B[i]);
        if(i % K == K - 1)
            printf("\n");
    }
    #endif
    float *d_A, *d_B, *d_C;
    hipMalloc((void**) &d_A, sizeof(float) * N * M);
    hipMalloc((void**) &d_B, sizeof(float) * M * K);
    hipMalloc((void**) &d_C, sizeof(float) * N * K);

    printf("start copying data from Host to Device\n");

    hipMemcpy(d_A, A, sizeof(float) * N * M, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeof(float) * M * K, hipMemcpyHostToDevice);
    
    printf("start calculating\n");
    hipEvent_t e_start, e_stop;
    hipEventCreate(&e_start);
    hipEventCreate(&e_stop);

    hipEventRecord(e_start, 0);
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, K, M, N, &alpha, d_B, K, d_A, N, &beta, d_C, K);
    hipblasDestroy(handle);
    hipEventRecord(e_stop, 0);

    hipEventSynchronize(e_stop);
    float elapsedtime;
    hipEventElapsedTime(&elapsedtime, e_start, e_stop);
    printf("finish calculating, costing %f\n", elapsedtime);
    
    printf("start copying data from Device to Host\n");
    hipMemcpy(C, d_C, sizeof(float) * N * K, hipMemcpyDeviceToHost);
    printf("Normal execution\n");
    #ifdef DEBUG
    for(int i = 0; i < N * K; ++ i) {
        printf("%f ",C[i]);
        if(i % K == K - 1)
            printf("\n");
    }
    #endif
    hipError_t err = hipGetLastError();
    printf("%s\n",hipGetErrorString(err));
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);
}
