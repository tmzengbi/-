#ifndef __CUDACC__
#define __CUDACC__
#endif

#include <cstdio>
#include <cstring>
#include <hip/hip_runtime.h>
#include <hipblas.h>
using namespace std;

#define stride 1
#define THREADS_PER_BLOCK 32


__global__ void cudacnn(float *input, float *kernel, float *out, int N, int M, int K) {
    int threadId_x = blockIdx.x * blockDim.x + threadIdx.x;
    int threadId_y = blockIdx.y * blockDim.y + threadIdx.y;
    if(threadId_x >= K || threadId_y >= K)
        return;
    // printf("%d %d\n", threadId_x, threadId_y);
    int x = threadId_x * stride, y = threadId_y * stride;
    float sum = 0;
    for(int i = 0; i < M ; ++ i) 
        for(int j = 0; j < M ; ++ j) {
            if(x + i < N && y + j < N) 
                sum += input[(x + i) * N + (y + j)] * kernel[i * M + j];
        }
    out[threadId_x * K + threadId_y] = sum;
}   

int main(int argc, char *argv[]) {
    float *input, *kernel, *output;
    int N, M, K;
    FILE *fp;
    if((fp = fopen("a.in", "r")) == NULL) {
        perror("cannot open file");
        exit(-1);
    }
    fread(&N, 1, sizeof(int), fp);
    fread(&M, 1, sizeof(int), fp);
    // K = (N - M + 1 + stride - 1) / stride
    K = (N - M + stride) / stride;
    input = (float*) malloc (sizeof(float) * N * N);
    kernel = (float*) malloc (sizeof(float) * M * M);
    output = (float*) malloc (sizeof(float) * K * K);
    fread(input, sizeof(float), N * N, fp);
    fread(kernel, sizeof(float), M * M, fp);
    #ifdef DEBUG
    printf("Matrix input:\n");
    for(int i = 0; i < N * N; ++ i) {
        printf("%f ",input[i]);
        if(i % N == N - 1)
            printf("\n");
    }
    printf("Matrix kernel:\n");
    for(int i = 0; i < M * M; ++ i) {
        printf("%f ",kernel[i]);
        if(i % M == M - 1)
            printf("\n");
    }
    #endif
    float *d_input, *d_kernel, *d_output;
    hipMalloc((void**) &d_input, sizeof(float) * N * N);
    hipMalloc((void**) &d_kernel, sizeof(float) * M * M);
    hipMalloc((void**) &d_output, sizeof(float) * K * K);

    printf("start copying data from Host to Device\n");

    hipMemcpy(d_input, input, sizeof(float) * N * N, hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel, sizeof(float) * M * M, hipMemcpyHostToDevice);
    
    printf("start calculating\n");
    hipEvent_t e_start, e_stop;
    hipEventCreate(&e_start);
    hipEventCreate(&e_stop);

    hipEventRecord(e_start, 0);
    
    dim3 block((K + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, (K + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);
    dim3 threads(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    cudacnn<<<block, threads>>>(d_input, d_kernel, d_output, N, M, K);

    hipEventRecord(e_stop, 0);

    hipEventSynchronize(e_stop);
    float elapsedtime;
    hipEventElapsedTime(&elapsedtime, e_start, e_stop);
    printf("finish calculating, costing %f\n", elapsedtime);
    
    printf("start copying data from Device to Host\n");
    hipMemcpy(output, d_output, sizeof(float) * K * K, hipMemcpyDeviceToHost);
    printf("Normal execution\n");
    #ifdef DEBUG
    for(int i = 0; i < K * K; ++ i) {
        printf("%f ",output[i]);
        if(i % K == K - 1)
            printf("\n");
    }
    #endif
    hipError_t err = hipGetLastError();
    printf("%s\n",hipGetErrorString(err));
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_kernel);
}
